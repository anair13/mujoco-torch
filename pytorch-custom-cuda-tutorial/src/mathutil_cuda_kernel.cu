#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <stdio.h>
#include <math.h>
#include <float.h>

#include "mathutil_cuda_kernel.h"

dim3 cuda_gridsize(int n)
{
    int k = (n - 1) / BLOCK + 1;
    int x = k;
    int y = 1;
    if(x > 65535) {
        x = ceil(sqrt(k));
        y = (n - 1) / (x * BLOCK) + 1;
    }
    dim3 d(x, y, 1);
    return d;
}

__global__ void broadcast_sum_kernel(float *a, float *b, int x, int y, int size)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= size) return;
    int j = i % y; i = i / y;
    int k = i % x;
    a[IDX2D(k, j, y)] += b[k];
}

void broadcast_sum_cuda(float *a, float *b, int x, int y, hipStream_t stream)
{
    int size = x * y;
    hipError_t err;

    broadcast_sum_kernel<<<cuda_gridsize(size), BLOCK, 0, stream>>>(a, b, x, y, size);

    err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
